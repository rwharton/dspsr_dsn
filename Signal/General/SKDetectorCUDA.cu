#include "hip/hip_runtime.h"
//-*-C++-*-

/***************************************************************************
 *
 *   Copyright (C) 2016 by Andrew Jameson
 *   Licensed under the Academic Free License version 2.1
 *
 ***************************************************************************/


#include "dsp/SKDetectorCUDA.h"

#include <iostream>
#include <thrust/system/cuda/execution_policy.h>
#include <thrust/reduce.h>
#include <thrust/device_ptr.h>

#include <hip/hip_complex.h>

#define FULL_MASK 0xffffffff

//#define _DEBUG 1

// TODO consider having schan / echan in mask represented by values other than 0, 1

using namespace std;

void check_error (const char*);

CUDA::SKDetectorEngine::SKDetectorEngine (dsp::Memory * memory)
{
  device_memory = dynamic_cast<CUDA::DeviceMemory *>(memory);
  stream = device_memory->get_stream();

  estimates_host = new dsp::TimeSeries();
  zapmask_host = new dsp::BitSeries();

  pinned_memory  = new PinnedMemory ();
  estimates_host->set_memory ((dsp::Memory *) pinned_memory);
  zapmask_host->set_memory ((dsp::Memory *) pinned_memory);

  transfer_estimates = new dsp::TransferCUDA (stream);
  transfer_estimates->set_kind (hipMemcpyDeviceToHost);
  transfer_estimates->set_output( estimates_host );

  transfer_zapmask = new dsp::TransferBitSeriesCUDA (stream);
  transfer_zapmask->set_kind (hipMemcpyDeviceToHost);
  transfer_zapmask->set_output( zapmask_host );
}

void CUDA::SKDetectorEngine::setup ()
{
  if (dsp::Operation::verbose)
    cerr << "CUDA::SKDetectorEngine::setup ()" << endl;

  // determine GPU capabilities
  int device = 0;
  hipGetDevice(&device);
  struct hipDeviceProp_t device_properties;
  hipGetDeviceProperties (&device_properties, device);
  max_threads_per_block = device_properties.maxThreadsPerBlock;
}


// faster kernel for npol=1
__global__ void detect_one_pol (const float * indat, unsigned char * outdat, uint64_t nval, float upper, float lower)
{
  unsigned idat  = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (idat < nval)
  {
    float V = indat[idat];
    if (V < lower || V > upper)
      outdat[idat] = 1;
  }
}

__global__ void detect_two_pol (const float2 * indat, unsigned char * outdat, uint64_t nval, float upper, float lower)
{
  unsigned idat  = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (idat < nval)
  {
    const float2 V = indat[idat];
    if (V.x < lower || V.x > upper || V.y < lower || V.y > upper)
    {
      outdat[idat] = 1;
    }
  }
}


// detect SK limits for N polarisations
__global__ void detect_one_sample (const float * indat, unsigned char * outdat, uint64_t nval, float upper, float lower, unsigned npol)
{
  unsigned idat  = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (idat < nval)
  {
    unsigned zap = 0;
    float V;

    for (int ipol=0; ipol<npol; ipol++)
    {
      V = indat[(idat * npol) + ipol];
      if (V < lower || V > upper)
      {
        zap = 1;
      }
    }
    if (zap)
      outdat[idat] = 1;
  }
}

void CUDA::SKDetectorEngine::detect_ft (const dsp::TimeSeries* input,
      dsp::BitSeries* output, float upper_thresh, float lower_thresh)
{
  if (dsp::Operation::verbose)
    cerr << "CUDA::SKDetectorEngine::detect_ft()" << endl;

  const unsigned nchan = input->get_nchan();
  const unsigned npol  = input->get_npol();
  const int64_t  ndat  = input->get_ndat();

  const float * indat    = input->get_dattfp();   // TFP
  unsigned char * outdat = output->get_datptr();  // TFP also!

  uint64_t nval   = nchan * ndat;
  uint64_t nblocks  = nval / max_threads_per_block;
  if (nval % max_threads_per_block)
    nblocks++;

  dim3 threads (max_threads_per_block);
  dim3 blocks (nblocks);

  if (dsp::Operation::verbose)
  {
    cerr << "CUDA::SKDetectorEngine::detect_ft nval=" << nval << " nblocks=" << nblocks << " max_threads_per_block=" << max_threads_per_block << endl;
    cerr << "CUDA::SKDetectorEngine::detect_ft thresholds [" << lower_thresh << " - " << upper_thresh << "]" << endl;
    cerr << "CUDA::SKDetectorEngine::detect_ft npol=" << npol << endl;
  }

  if (npol == 1)
    detect_one_pol<<<blocks,threads,npol,stream>>> (indat, outdat, nval, upper_thresh, lower_thresh);
  else if (npol == 2)
    detect_two_pol<<<blocks,threads,npol,stream>>> ((const float2 *) indat, outdat, nval, upper_thresh, lower_thresh);
  else
    detect_one_sample<<<blocks,threads,npol,stream>>> (indat, outdat, nval, upper_thresh, lower_thresh, npol);

  if (dsp::Operation::record_time || dsp::Operation::verbose)
    check_error( "CUDA::SKDetectorEngine::detect_ft detect_one_xxx" );

#ifdef _DEBUG
  int sum = count_mask(output);
  cerr << "CUDA::SKDetectorEngine::detect_ft sum now " << sum << endl;
#endif
}

__device__ float2 warp_reduce_sum (float2 val) {
  for (int offset = warpSize/2; offset > 0; offset >>= 1) {
    #if (__CUDACC_VER_MAJOR__>= 9)
    val.x += __shfl_down_sync(FULL_MASK, val.x, offset);
    val.y += __shfl_down_sync(FULL_MASK, val.y, offset);
    #else
    val.x += __shfl_down (val.x, offset);
    val.y += __shfl_down (val.y, offset);
    #endif
  }
  return val;
}

__device__ float3 warp_reduce_sum (float3 val) {
  for (int offset = warpSize/2; offset > 0; offset >>= 1) {
    #if (__CUDACC_VER_MAJOR__>= 9)
    val.x += __shfl_down_sync(FULL_MASK, val.x, offset);
    val.y += __shfl_down_sync(FULL_MASK, val.y, offset);
    val.z += __shfl_down_sync(FULL_MASK, val.z, offset);
    #else
    val.x += __shfl_down (val.x, offset);
    val.y += __shfl_down (val.y, offset);
    val.z += __shfl_down (val.z, offset);
    #endif
  }
  return val;
}





// each block reads 1 time sample, all channels/pols
// then do a block-wide sum

// input data are stored TFP, 1 warp per time sample, 32 warps / block to sum across channels
// __global__ void reduce_sum_fscr_1pol (const float * input, unsigned char * out,
//                                       const unsigned nchan, float lower, float upper,
//                                       unsigned schan, unsigned echan)
// {
//   extern __shared__ float sdata[];
//
//   unsigned idat = blockIdx.x;
//   const float * in = input + (idat * nchan);
//
//   float sum = 0;
//   for (unsigned ichan=threadIdx.x; ichan<nchan; ichan+=blockDim.x)
//   {
//     if (ichan >= schan && ichan < echan)
//       sum += in[ichan];
//   }
//
//   sdata[threadIdx.x] = sum;
//   __syncthreads();
//
//   // now do a block wide sum across all threads
//   int last_offset = blockDim.x / 2 ;
//   for (int offset = last_offset; offset > 0;  offset >>= 1)
//   {
//     if (threadIdx.x < offset)
//       sdata[threadIdx.x] += sdata[threadIdx.x + offset];
//
//     __syncthreads();
//   }
//
//   if (threadIdx.x == 0)
//   {
//     float val = sdata[0] / float((echan - schan) + 1);
//     if (val < lower || val > upper)
//       out[idat] = 1;
//   }
// }

__global__ void reduce_sum_fscr_1pol (
  const float * input, unsigned char * out,
  const unsigned nchan, const float mu2, const float std_devs,
  const unsigned schan, const unsigned echan
)
{
  extern __shared__ float2 sdata2[]; // we have nchan * (npol + 1) * sizeof(float) available bytes

  // idat = blockIdx.x
  // use float 2 because input is TFP, meaning we can bundle polarizations
  // as if they were complex number
  const float * in = input + (blockIdx.x * nchan);

  float2 sum = make_float2(0, 0);
  for (unsigned ichan=threadIdx.x; ichan<nchan; ichan+=blockDim.x)
  {
    if (ichan >= schan && ichan < echan && out[blockIdx.x * nchan + ichan] == 0) {
      sum.x += in[ichan];
      sum.y += 1;
    }
  }

  sum = warp_reduce_sum(sum);

  unsigned warp_idx = threadIdx.x % 32;
  unsigned warp_num = threadIdx.x / 32;
  unsigned max_warp_num = blockDim.x / warpSize;


  if (warp_idx == 0) {
    sdata2[warp_num] = sum;
  }
  __syncthreads();

  if (warp_num == 0) {
    if (warp_idx >= max_warp_num) {
      sum = make_float2(0, 0);
    } else {
      sum = sdata2[warp_idx];
    }
    sum = sdata2[warp_idx];
    sum = warp_reduce_sum(sum);

    if (warp_idx == 0) {
      float sk_avg_cnt = sum.y;
      float one_sigma_idat = sqrtf(mu2 / (float) sk_avg_cnt);
      float p0 = sum.x / sk_avg_cnt;
      float upper = 1 + ((1+std_devs) * one_sigma_idat);
      float lower = 1 - ((1+std_devs) * one_sigma_idat);
      // printf("reduce_sum_fscr_2pol: p0=%f, p1=%f, lower=%f, upper=%f, sk_avg_cnt=%f, pol0 sum=%f, pol1 sum=%f\n", p0, p1, lower, upper, sk_avg_cnt, p0*sk_avg_cnt, p1*sk_avg_cnt);
      if (p0 < lower || p0 > upper) {
        sdata2[0].x = 1.0;
      } else {
        sdata2[0].x = 0.0;
      }
    }
  }

  __syncthreads();

  if (sdata2[0].x == 1.0) {
    for (unsigned ichan=threadIdx.x; ichan<nchan; ichan+=blockDim.x) {
      out[blockIdx.x * nchan + ichan] = 1;
    }
  }
}



//! @method reduce_sum_fscr_2pol Determine whether SK statistic is outside
//! of bounds for two polarization input data.
//! blockDim.x is nchan, so threadIdx.x is ichan
//! gridDim.x is input->get_ndat(), or npart, os blockIdx.x is ipart
//! This kernel packs polarization data into float3 data.
//! @param input (npart, nchan, npol) TFP ordered SK statistic
//! @param out (npart, nchan, 1) TFP ordered zapmask
//! @param nchan number of channels present in data
//! @param mu2 value used to calculate bounds for SK statistic
//! @param std_devs number of standard deviations outside of which SK statistic
//!   will be rejected
//! @param schan start channel. Defines the start of the frequency domain
//!   analysis region
//! @param echan end channel. Defines the end of the frequency domain
//!   analysis region
__global__ void reduce_sum_fscr_2pol (
  const float2 * input, unsigned char * out,
  const unsigned nchan, const float mu2, const float std_devs,
  const unsigned schan, const unsigned echan
)
{
  extern __shared__ float3 sdata3[]; // we have nchan * (npol + 1) * sizeof(float) available bytes

  // idat = blockIdx.x
  // use float 2 because input is TFP, meaning we can bundle polarizations
  // as if they were complex number
  const float2 * in = input + (blockIdx.x * nchan);

  float3 sum = make_float3(0, 0, 0);
  for (unsigned ichan=threadIdx.x; ichan<nchan; ichan+=blockDim.x)
  {
    if (ichan >= schan && ichan < echan && out[blockIdx.x * nchan + ichan] == 0) {
      sum.x += in[ichan].x;
      sum.y += in[ichan].y;
      sum.z += 1.0;
    }
  }
  sum = warp_reduce_sum(sum);

  unsigned warp_idx = threadIdx.x % warpSize;
  unsigned warp_num = threadIdx.x / warpSize;
  unsigned max_warp_num = blockDim.x / warpSize;

  if (warp_idx == 0) {
    sdata3[warp_num] = sum;
  }
  __syncthreads();

  if (warp_num == 0) {
    if (warp_idx >= max_warp_num) {
      sum = make_float3(0, 0, 0);
    } else {
      sum = sdata3[warp_idx];
    }
    sum = warp_reduce_sum(sum);

    if (warp_idx == 0) {
      float sk_avg_cnt = sum.z;
      float one_sigma_idat = sqrtf(mu2 / sk_avg_cnt);
      float p0 = sum.x / sk_avg_cnt;
      float p1 = sum.y / sk_avg_cnt;
      float upper = 1 + ((1+std_devs) * one_sigma_idat);
      float lower = 1 - ((1+std_devs) * one_sigma_idat);
      if (p0 < lower || p0 > upper || p1 < lower || p1 > upper) {
        // printf("Zapping ipart=%u p0=%f, p1=%f [%f - %f] cnt=%f\n",
        //   blockIdx.x, p0, p1, lower, upper, sk_avg_cnt);
        // out[blockIdx.x] = 1;
        // for (unsigned ichan=0; ichan < nchan; ichan++) {
        //   out[blockIdx.x * nchan + ichan] = 1;
        // }
      }
      // printf("reduce_sum_fscr_2pol: p0=%f, p1=%f, lower=%f, upper=%f, sk_avg_cnt=%f, pol0 sum=%f, pol1 sum=%f\n", p0, p1, lower, upper, sk_avg_cnt, p0*sk_avg_cnt, p1*sk_avg_cnt);
      if (p0 < lower || p0 > upper || p1 < lower || p1 > upper) {
        sdata3[0].x = 1.0;
      } else {
        sdata3[0].x = 0.0;
      }
    }
  }

  __syncthreads();

  if (sdata3[0].x == 1.0) {
    for (unsigned ichan=threadIdx.x; ichan<nchan; ichan+=blockDim.x) {
      out[blockIdx.x * nchan + ichan] = 1;
    }
  }
}

// schan is the start channel and echan is the end channel. Together these
// define a range of channels that will be zapped.
// input is the TFP ordered SK estimates, of size (npart, nchan, npol)
// output is the TFP ordered zapmask, of size (npart, nchan, 1)
// Here, npart is the original TimeSeries input ndat divided by ``M``
void CUDA::SKDetectorEngine::detect_fscr (
  const dsp::TimeSeries* input, dsp::BitSeries* output,
  const float mu2, const float std_devs,
  unsigned schan, unsigned echan)
{
  if (dsp::Operation::verbose) {
    std::cerr << "CUDA::SKDetectorEngine::detect_fscr()" << std::endl;
  }

  const unsigned nchan = input->get_nchan();
  const unsigned npol = input->get_npol();
  const int64_t ndat = input->get_ndat();

  const unsigned nblocks = ndat;
  unsigned nthreads = max_threads_per_block;
  if (nchan < nthreads)
    nthreads = nchan;
  // const size_t shared_bytes = nthreads * (npol + 1) * sizeof(float);
  const size_t shared_bytes = 32 * (npol + 1) * sizeof(float);

  // indat is the SK estimates
  const float * indat = input->get_dattfp();

  // outdat is the bitmask
  unsigned char * outdat = output->get_datptr();

  if (dsp::Operation::verbose)
  {
    std::cerr << "CUDA::SKDetectorEngine::detect_fscr" <<
      " output->get_ndat()=" << output->get_ndat() <<
      " output->get_nchan()=" << output->get_nchan() <<
      " output->get_npol()=" << output->get_npol() << std::endl;

    std::cerr << "CUDA::SKDetectorEngine::detect_fscr nchan=" << nchan << " ndat=" << ndat << std::endl;
    std::cerr << "CUDA::SKDetectorEngine::detect_fscr nblocks=" << nblocks << " nthreads=" << nthreads << " shared_bytes=" << shared_bytes << std::endl;
  }

  if (npol == 1) {
    reduce_sum_fscr_1pol<<<nblocks, nthreads, shared_bytes, stream>>>(
      indat, outdat, nchan, mu2, std_devs, schan, echan);
  } else {
    reduce_sum_fscr_2pol<<<nblocks, nthreads, shared_bytes, stream>>>(
      (float2*) indat, outdat, nchan, mu2, std_devs, schan, echan);
  }

  if (dsp::Operation::record_time || dsp::Operation::verbose) {
    check_error( "CUDA::SKDetectorEngine::detect_fscr_element" );
  }

#ifdef _DEBUG
  int sum = count_mask(output);
  cerr << "CUDA::SKDetectorEngine::detect_fscr mask_sum=" << sum << endl;
#endif

  if (dsp::Operation::record_time || dsp::Operation::verbose) {
    check_error( "CUDA::SKDetectorEngine::detect detect_fscr" );
  }
}

// nval is output->get_ndat() * nchan
// indat is TFP ordered
// indat is (1, nchan, npol)
// outdat is TFP ordered
// outdat is (npart, nchan, 1)
__global__ void detect_tscr_element (
  const float * indat,
  unsigned char * outdat,
  const uint64_t nval,
  const float upper,
  const float lower,
  const unsigned npol,
  const unsigned nchan
)
{

  extern __shared__ char sk_tscr[];

  const unsigned idat  = (blockIdx.x * blockDim.x + threadIdx.x);
  // if (idat ==0) {
  //   printf("detect_tscr_element: npol=%u, nchan=%u\n", npol, nchan);
  // }
  if (idat < nval)
  {
    // const unsigned nchanpol = nchan * npol;
    // const unsigned ichanpol = idat % nchanpol;

    // first nchan threads to fill shared mem with the tscr SK estimates for each chan & pol (TFP)

    // if (threadIdx.x < nchan)
    // {
    //   // sk_tscr[threadIdx.x] = (char) (indat[threadIdx.x] > upper || indat[threadIdx.x] < lower);
    //   all_pol_in_thresh = false;
    //   for (unsigned ipol=0; ipol<npol; ipol++) {
    //     all_pol_in_thresh = all_pol_in_thresh || (indat[threadIdx.x*npol + ipol] > upper || indat[threadIdx.x*npol + ipol] < lower);
    //   }
    //   sk_tscr[threadIdx.x] = (char) all_pol_in_thresh;
    // }
    // __syncthreads();
    // outdat[idat/npol] = sk_tscr[ichanpol];
    const unsigned ichan = idat % nchan;
    if (ichan < nchan)
    {
      bool all_pol_in_thresh = false;
      for (unsigned ipol=0; ipol<npol; ipol++) {
        all_pol_in_thresh = (all_pol_in_thresh ||
          (indat[ichan*npol + ipol] > upper ||
           indat[ichan*npol + ipol] < lower));
      }
      sk_tscr[ichan] = (char) all_pol_in_thresh;
    }
    __syncthreads();
    outdat[idat] = sk_tscr[ichan];
  }
}


void CUDA::SKDetectorEngine::detect_tscr (
  const dsp::TimeSeries* input,
  const dsp::TimeSeries* input_tscr,
  dsp::BitSeries* output,
  float upper_thresh,
  float lower_thresh//,
  // unsigned schan,
  // unsigned echan
)
{
  if (dsp::Operation::verbose) {
    cerr << "CUDA::SKDetectorEngine::detect_tscr()" << endl;
  }

  const unsigned nchan = input->get_nchan();
  const unsigned npol = input->get_npol();
  const int64_t ndat = output->get_ndat();

  // indat is the tscr mask [nchan vals]
  const float * indat = input_tscr->get_dattfp();

  // outdat is the bitmask
  unsigned char * outdat = output->get_datptr();

  // this kernel is indexed on output rather than input
  const uint64_t nval = ndat * nchan;
  uint64_t nblocks  = nval / max_threads_per_block;
  if (nval % max_threads_per_block) {
    nblocks++;
  }
  dim3 threads (max_threads_per_block);
  dim3 blocks (nblocks);
  unsigned shared_bytes = nchan*sizeof(char);

  if (dsp::Operation::verbose) {
    std::cerr << "CUDA::SKDetectorEngine::detect_tscr_element ndat=" << ndat
        << " npol=" << npol
         << " nchan=" << nchan << " nval=" << nval
         << " max_threads=" << max_threads_per_block
         << " nblocks=" << nblocks << std::endl;
  }
  detect_tscr_element<<<blocks, threads, shared_bytes, stream>>>(
    indat, outdat, nval, upper_thresh, lower_thresh, npol, nchan);

  if (dsp::Operation::record_time || dsp::Operation::verbose) {
    check_error( "CUDA::SKDetectorEngine::detect_tscr_element" );
  }
#ifdef _DEBUG
  int sum = count_mask(output);
  cerr << "CUDA::SKDetectorEngine::detect_tscr mask_sum=" << sum << endl;
#endif
}


void CUDA::SKDetectorEngine::reset_mask (dsp::BitSeries* output)
{
  unsigned nchan         = output->get_nchan();
  int64_t ndat           = output->get_ndat();
  unsigned char * outdat = output->get_datptr();

  size_t nbytes = nchan * ndat;

  hipError_t error = hipMemsetAsync (outdat, 0, nbytes, stream);
  if (error != hipSuccess)
    throw Error (FailedCall, "CUDA::SKDetectorEngine::reset_mask ",
                 "hipMemset (%p, 0, %u): %s", outdat, nbytes,
                 hipGetErrorString (error));
#ifdef _DEBUG
  int sum = count_mask(output);
  cerr << "CUDA::SKDetectorEngine::reset_mask sum now " << sum << endl;
#endif
}

int CUDA::SKDetectorEngine::count_mask (const dsp::BitSeries* output)
{
  unsigned char * outdat = const_cast<unsigned char *>(output->get_datptr());
  const unsigned nchan   = output->get_nchan();
  const int64_t ndat     = output->get_ndat();
  int sum = 0;
/*
  const uint64_t nval    = (uint64_t) ndat * nchan;
  hipStreamSynchronize(stream);
  thrust::device_ptr<unsigned char> d = thrust::device_pointer_cast(outdat);
  int sum = thrust::reduce(thrust::cuda::par.on(stream), d, d+nval, (int) 0, thrust::plus<int>());
  hipStreamSynchronize(stream);
*/

  return sum;
}

float * CUDA::SKDetectorEngine::get_estimates (const dsp::TimeSeries * input)
{
  transfer_estimates->set_input (input);
  transfer_estimates->operate ();
  hipStreamSynchronize (stream);
  return estimates_host->get_dattfp();
}

unsigned char * CUDA::SKDetectorEngine::get_zapmask (const dsp::BitSeries * input)
{
  transfer_zapmask->set_input (input);
  transfer_zapmask->operate ();
  hipStreamSynchronize (stream);
  return zapmask_host->get_datptr();
}
